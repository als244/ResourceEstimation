
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblaslt.h>
#include <hiprand/hiprand.h>
#include <time.h>


int main(int argc, char * argv[]){

	if (argc != 4){
		fprintf(stderr, "Wrong number of args\n");
		exit(1);
	}

	int m = atoi(argv[1]);
	int k = atoi(argv[2]);
	int n = atoi(argv[3]);

	if (m <= 0 || k <= 0 || n <= 0){
		fprintf(stderr, "Bad dimensions\n");
		exit(1);
	}

	// using cuRand to populate matrices
	hiprandGenerator_t gen;
	hiprandStatus_t curand_status;
	curand_status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	curand_status = hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

	// Init Matrices
	float * A, *B, *D;
	hipMalloc(&A, m * k * sizeof(float));
	hipMalloc(&B, k * n * sizeof(float));
	hipMalloc(&D, m * n * sizeof(float));

	// choosing arbitrary stddev that might resemeble init weights of Neural Network
	curand_status = hiprandGenerateNormal(gen, A, (size_t) (m * k), 0, sqrtf(1.0 / 20.0));
	curand_status = hiprandGenerateNormal(gen, B, (size_t) (k * n), 0, sqrtf(1.0 / 20.0));

	curand_status = hiprandDestroyGenerator(gen);

	// PERFORM MATMUL

	hipblasStatus_t status;
	hipblasLtHandle_t handle;
	status = hipblasLtCreate(&handle);


	hipblasOperation_t transa = HIPBLAS_OP_T;
	hipblasOperation_t transb = HIPBLAS_OP_N;

	hipblasLtMatrixLayout_t Adesc;
	hipblasLtMatrixLayout_t Bdesc;
	hipblasLtMatrixLayout_t Cdesc;
	hipblasLtMatrixLayout_t Ddesc;


	hipblasLtMatmulDesc_t matmulDesc;

	status = hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F_FAST_16F, HIP_R_32F);
	status = hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
	status = hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));

	// A Transposed (from row-major to column-major), not B/D (but still held in col-major format internally)
	// m and k must be multiples of 4, perferablly multiples of 16
	status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, k, m, k);
	status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, k, n, k);
	status = hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, m);
	status = hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, m, n, m);


	hipblasLtMatmulPreference_t pref;
	status = hipblasLtMatmulPreferenceCreate(&pref);
	// ALLOW 1 GB of workspace mem...
	//const size_t workspaceBytes = 1000000000;
	const size_t workspaceBytes = 0;
	//status = cublasLtMatmulPreferenceSetAttribute(pref, CUBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceBytes, sizeof(workspaceBytes));

	int algoCount = 1;
	int retAlgoCount = 0;

	hipblasLtMatmulHeuristicResult_t heuristicResultsArray = {};


	status = hipblasLtMatmulAlgoGetHeuristic(handle, matmulDesc, Adesc, Bdesc, Cdesc, Ddesc, pref, algoCount, &heuristicResultsArray, &retAlgoCount);

	hipblasLtMatmulAlgo_t algo = heuristicResultsArray.algo;

	//void * workspace;
	void * workspace = NULL;
	//cudaMalloc(&workspace, workspaceBytes);

	float alpha = 1, beta = 0;
	
	status = hipblasLtMatmul(handle,
							matmulDesc,
							&alpha,
							A,
							Adesc,
							B,
							Bdesc,
							&beta,
							NULL,
							Cdesc,
							D,
							Ddesc,
							&algo,
							workspace,
							workspaceBytes,
							0);
	
	
	// FREE workspace
	hipFree(workspace);


	// FREE cuBlasLt Structs 

	status = hipblasLtMatmulPreferenceDestroy(pref);
	status = hipblasLtMatmulDescDestroy(matmulDesc);

	status = hipblasLtMatrixLayoutDestroy(Adesc);
	status = hipblasLtMatrixLayoutDestroy(Bdesc);
	status = hipblasLtMatrixLayoutDestroy(Cdesc);
	status = hipblasLtMatrixLayoutDestroy(Ddesc);

	status = hipblasLtDestroy(handle);


	// FREE MATRICES

	hipFree(A);
	hipFree(B);
	hipFree(D);

}